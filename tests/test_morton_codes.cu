#include "hip/hip_runtime.h"
//
// Created by Erin M Gunn on 1/17/21.
//

#include <cassert>
#include <iostream>
#include <iomanip>

#include "../radtree.h"

int test_mc(const uint32_t mc, const float3 point) {
    if (mc != radtree::utils::encode_morton_code(point)) {
        std::cout << "Encoding Failed\n";
        std::cout << "    Expected: 0x" << std::hex << std::setw(8) << std::setfill('0') << mc << '\n';
        auto actual = radtree::utils::encode_morton_code(point);
        std::cout << "    Actual: 0x" << std::hex << std::setw(8) << std::setfill('0') << actual << '\n';
        return 1;
    }
    return 0;
};

int main(void) {
    int out = 0;

    out += test_mc(0x00000000, make_float3(0.999999f, 0.99999f, 0.99999f));
    out += test_mc(0x00000001, make_float3(1.0f, 0.0f, 0.0f));
    out += test_mc(0x00000002, make_float3(0.0f, 1.0f, 0.0f));
    out += test_mc(0x00000003, make_float3(1.0f, 1.0f, 0.0f));
    out += test_mc(0x00000004, make_float3(0.0f, 0.0f, 1.0f));

    out += test_mc(1 << 29, make_float3(0.0f, 0.0f, 512.0f));
    out += test_mc(0x3FFFFFFF, make_float3(1024.0f, 1024.0f, 1024.0f));

    return out;
}