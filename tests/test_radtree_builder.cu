#include "hip/hip_runtime.h"
//
// Created by Erin M Gunn on 1/17/21.
//

#include "../radtree.h"
#include "../timer.h"

#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main(void) {
    TIMER::START("Generating 32M random float3");
    thrust::host_vector<float3> h_vec(32 << 20);
    std::generate(h_vec.begin(), h_vec.end(), []() -> float3 { return { (float)rand(), (float)rand(), (float)rand() }; });
    TIMER::STOP();

    TIMER::START("Transfering data to device");
    thrust::device_vector<float3> d_vec = h_vec;
    TIMER::STOP();

    TIMER::START("Building radtree");
    radtree::builder b;
    auto rt = b.build(d_vec.begin(), d_vec.end());
    TIMER::STOP();
}