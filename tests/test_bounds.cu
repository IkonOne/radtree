#include "hip/hip_runtime.h"
//
// Created by Erin M Gunn on 1/16/21.
//

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <algorithm>
#include "radtree/radtree.h"
#include "../timer.h"

int main(void)
{
    std::cout << "sizeof(float3) = " << sizeof(float3) << " Bytes \n";
    std::cout << "sizeof(float3) * 32M = " << sizeof(float3) * (32 << 20) << " Bytes \n";
    std::cout << sizeof(float3) * 32 << " MB \n";

    TIMER::START("Generating 32M random float3");
    thrust::host_vector<float3> h_vec(32 << 20);
    std::generate(h_vec.begin(), h_vec.end(), []() -> float3 { return { (float)rand(), (float)rand(), (float)rand() }; });
    TIMER::STOP();

    TIMER::START("Transfering data to device");
    thrust::device_vector<float3> d_vec = h_vec;
    TIMER::STOP();

    TIMER::START("Reducing a bounding box containing all of the random float3's");
    auto b = radtree::utils::bounds::from_device(d_vec.begin(), d_vec.end());
    TIMER::STOP();

    std::cout << '\n';
    std::cout << "Bounds\n";
    std::cout << "Min: " << b.min.x << " " << b.min.y << " " << b.min.z << '\n';
    std::cout << "Max: " << b.max.x << " " << b.max.y << " " << b.max.z << '\n';
    std::cout << '\n';

    // transfer data back to host
    TIMER::START("Transfer data to host");
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
    TIMER::STOP();

    return 0;
}
